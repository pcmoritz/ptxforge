// Driver to execute a PTX program from the host

#include <iostream>
#include <hip/hip_runtime.h>

// Error checking macro
#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error: " << err << " in line " << __LINE__ << std::endl; \
        exit(-1); \
    }

int main() {
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;
    hipFunction_t cuFunction;

    // Initialize the CUDA driver API
    CUDA_CHECK(hipInit(0));

    // Get the first CUDA device
    CUDA_CHECK(hipDeviceGet(&cuDevice, 0));

    // Create a context on this device
    CUDA_CHECK(hipCtxCreate(&cuContext, 0, cuDevice));

    // Load the CUBIN module
    CUDA_CHECK(hipModuleLoad(&cuModule, "loop.cubin"));

    // Get the kernel function from the module
    CUDA_CHECK(hipModuleGetFunction(&cuFunction, cuModule, "main"));

    // Launch the kernel with a single block and single thread
    CUDA_CHECK(hipModuleLaunchKernel(cuFunction,
                              1, 1, 1,  // Grid dimensions
                              1, 1, 1,  // Block dimensions
                              0, 0,     // Shared memory and stream
                              NULL, 0)  // Kernel arguments
    );

    // Synchronize the device to wait for the kernel to finish
    CUDA_CHECK(hipCtxSynchronize());

    // Cleanup resources
    CUDA_CHECK(hipModuleUnload(cuModule));
    CUDA_CHECK(hipCtxDestroy(cuContext));

    std::cout << "Kernel execution completed." << std::endl;
    return 0;
}